#include "GEMM.cuh"
#include "GEMM_kernels.cuh"
#include "utils.cuh"
#include <chrono>
GEMM::~GEMM() {}

void GEMM::print_matrix(const float *matrix, int row, int col) {
  for (int i = 0; i < row; i++) {
    for (int j = 0; j < col; j++) {
      std::cout << matrix[col * i + j] << " ";
    }
    std::cout << "\n";
  }
}
void GEMM::run_tests() {
  // run helper functions to launch kernels
  naive_kernel(this->a, this->b, this->c, this->M, this->N, this->K,
               this->block_size);
  // print_matrix(this->c, this->M, this->N);
  eq_check(this->c, this->ref, this->M, this->N);
  mem_coalesce_kernel(this->a, this->b, this->c, this->M, this->N, this->K,
                      this->block_size);
  eq_check(this->c, this->ref, this->M, this->N);
  //                     this->block_size);
  // shared_mem_kernel(this->a, this->b, this->c, this->M, this->N, this->K,
  //                   this->block_size);
  // tiling_kernel(this->a, this->b, this->c, this->M, this->N, this->K,
  //               this->block_size);
  // tiling_kernel_v2(this->a, this->b, this->c, this->M, this->N, this->K,
  //                  this->block_size);
}

void GEMM::eq_check(const float *truth, const float *test, int row, int col) {
  float *d_truth;
  float *d_test;
  int *d_mismatch_flag;
  CUDA_CHECK(hipMalloc(&d_truth, sizeof(float) * row * col));
  CUDA_CHECK(hipMalloc(&d_test, sizeof(float) * row * col));
  CUDA_CHECK(hipMalloc(&d_mismatch_flag, sizeof(int)));

  int mismatch_flag = 0;
  CUDA_CHECK(hipMemcpy(d_mismatch_flag, &mismatch_flag, sizeof(int),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_truth, truth, sizeof(float) * row * col,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_test, test, sizeof(float) * row * col,
                        hipMemcpyHostToDevice));

  dim3 block_dim{32, 32, 1};
  dim3 grid_dim{(row + block_dim.x - 1) / block_dim.x,
                (col + block_dim.y - 1) / block_dim.y, 1};
  eqCheck<<<grid_dim, block_dim>>>(d_truth, d_test, row, col, d_mismatch_flag);

  CUDA_CHECK(hipMemcpy(&mismatch_flag, d_mismatch_flag, sizeof(int),
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipGetLastError());
  CUDA_CHECK(hipDeviceSynchronize());
  if (mismatch_flag == 1) {
    std::cout << "THEY DO NO MATCH\n";
  } else {
    std::cout << "YAYYY! they matched\n";
  }
  CUDA_CHECK(hipFree(d_mismatch_flag));
  CUDA_CHECK(hipFree(d_truth));
  CUDA_CHECK(hipFree(d_test));
}

void GEMM::naive_kernel(const float *a, const float *b, float *c, int M, int N,
                        int K, int block_size) {
  float *d_a;
  float *d_b;
  float *d_c;
  CUDA_CHECK(hipMalloc(&d_a, sizeof(float) * M * K));
  CUDA_CHECK(hipMalloc(&d_b, sizeof(float) * N * K));
  CUDA_CHECK(hipMalloc(&d_c, sizeof(float) * M * N));

  CUDA_CHECK(hipMemcpy(d_a, a, sizeof(float) * M * K, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, b, sizeof(float) * N * K, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_c, 0, sizeof(float) * M * N));
  dim3 block_dim{32, 32, 1};
  dim3 grid_dim{(M + block_dim.x - 1) / block_dim.x,
                (N + block_dim.y - 1) / block_dim.y, 1};
  // each thread will be responsible for a single output cell
  auto start = std::chrono::high_resolution_clock::now();
  naiveKernel<<<grid_dim, block_dim>>>(d_a, d_b, d_c, M, N, K);
  CUDA_CHECK(hipGetLastError());
  auto end = std::chrono::high_resolution_clock::now();
  const std::chrono::duration<double> duration = end - start;
  std::cout << "Naive Kernel finished with a latency of: " << duration.count()
            << " seconds!" << std::endl;

  CUDA_CHECK(hipMemcpy(c, d_c, sizeof(float) * M * N, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipFree(d_c));
}

void GEMM::mem_coalesce_kernel(const float *a, const float *b, float *c, int M,
                               int N, int K, int block_size) {
  float *d_a, *d_b, *d_c;

  CUDA_CHECK(hipMalloc(&d_a, sizeof(float) * M * K));
  CUDA_CHECK(hipMalloc(&d_b, sizeof(float) * N * K));
  CUDA_CHECK(hipMalloc(&d_c, sizeof(float) * M * N));

  CUDA_CHECK(hipMemcpy(d_a, a, sizeof(float) * M * K, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b, b, sizeof(float) * N * K, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemset(d_c, 0, sizeof(float) * M * N));
  dim3 block_dim{32, 32, 1};
  dim3 grid_dim{(M + block_dim.x - 1) / block_dim.x,
                (N + block_dim.y - 1) / block_dim.y, 1};
  memCoalesce<<<grid_dim, block_dim>>>(d_a, d_b, d_c, M, N, K);
  CUDA_CHECK(hipGetLastError());

  CUDA_CHECK(hipFree(d_a));
  CUDA_CHECK(hipFree(d_b));
  CUDA_CHECK(hipFree(d_c));
}

void GEMM::shared_mem_kernel(const float *a, const float *b, float *c, int M,
                             int N, int K, int block_size) {
  // TODO
}

void GEMM::tiling_kernel(const float *a, const float *b, float *c, int M, int N,
                         int K, int block_size) {
  // TODO
}

void GEMM::tiling_kernel_v2(const float *a, const float *b, float *c, int M,
                            int N, int K, int block_size) {
  // TODO
}
