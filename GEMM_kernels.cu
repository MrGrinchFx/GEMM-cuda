#include "hip/hip_runtime.h"

#include "GEMM_kernels.cuh"
#include <hip/hip_runtime.h>
void __global__ naiveKernel(const float *a, const float *b, float *c, int M,
                            int N, int K) {
  int cRow = blockDim.x * blockIdx.x + threadIdx.x;
  int cCol = blockDim.y * blockIdx.y + threadIdx.y;

  float result = 0.0f;
  if (cRow < M && cCol < N) {
    for (int i = 0; i < K; i++) {
      result += a[cRow * K + i] * b[N * i + cCol];
    }

    c[cRow * N + cCol] = result;
  }
}

void __global__ eqCheck(const float *truth, const float *test, int rows,
                        int cols, int *mismatchFlag) {
  int cRow = blockDim.x * blockIdx.x + threadIdx.x;
  int cCol = blockDim.y * blockIdx.y + threadIdx.y;

  if (cCol < cols && cRow < rows) {
    if (fabs(truth[cRow * cols + cCol] - test[cRow * cols + cCol]) > 0.01f) {
      atomicExch(mismatchFlag, 1);
    }
  }
}

// void __global__ memCoalesce(const float *a, const float *b, float *c,
//                             int size) {
//   // TODO
// }

void __global__ sharedMem(const float *a, const float *b, float *c, int size) {
  // TODO
}

void __global__ tiling2D(const float *a, const float *b, float *c, int size) {
  // TODO
}

void __global__ tiling2D_V2(const float *a, const float *b, float *c,
                            int size) {
  // TODO
}
