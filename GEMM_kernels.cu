
#include <hip/hip_runtime.h>

void __global__ naiveKernel(const float *a, const float *b, float *c,
                            int size) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx < size) {
    c[idx] = a[idx] * b[idx];
  }
}

void __global__ memCoalesce(const float *a, const float *b, float *c,
                            int size) {
  // TODO
}

void __global__ sharedMem(const float *a, const float *b, float *c, int size) {
  // TODO
}

void __global__ tiling2D(const float *a, const float *b, float *c, int size) {
  // TODO
}

void __global__ tiling2D_V2(const float *a, const float *b, float *c,
                            int size) {
  // TODO
}
